#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mathKernel1(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if (tid % 2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel2(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    // set up data size
    int size = 64;
    int blocksize = 64;
    if (argc > 1) blocksize = atoi(argv[1]);
    if (argc > 2) size = atoi(argv[2]);
    printf("Data size %d\n", size);

    // set up execution configuration
    dim3 block(blocksize, 1);
    dim3 grid((size+block.x-1)/block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**)&d_C, nBytes);

    // run a warmup kernel to remove overhead
    size_t iStart, iElaps;
    hipDeviceSynchronize();
    iStart = seconds();
    warmingup<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("warmup  <<<%4d %4d>>> elapsed %d sec \n", grid.x, block.x, iElaps);

    // run kernel1
    iStart = seconds();
    mathKernel1<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("mathKernel1  <<<%4d %4d>>> elapsed %d sec \n", grid.x, block.x, iElaps);

    // run kernel2
    iStart = seconds();
    mathKernel2<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("mathKernel2  <<<%4d %4d>>> elapsed %d sec \n", grid.x, block.x, iElaps);

    // free gpu memory and reset device
    hipFree(d_C);
    hipDeviceReset();
    return EXIT_SUCCESS;
}